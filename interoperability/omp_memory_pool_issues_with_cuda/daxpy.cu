#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.h"
#include "omp.h"
#include "hip/hip_runtime_api.h"

__global__
void daxpy_cudac(int n, double a, double *x, double *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void testDaxpy_cudac(void)
{
  size_t N = 12*(1<<26); // About 12GB
  double *x, *y, *d_x, *d_y;
  size_t free_bytes = 0, total_bytes = 0;
  hipError_t status;                                                                                                                                                                                             
   
  x = (double*)malloc(N*sizeof(double));
  y = (double*)malloc(N*sizeof(double));

  d_x = (double*)omp_target_alloc(N*sizeof(double), omp_get_default_device());
  d_y = (double*)omp_target_alloc(N*sizeof(double), omp_get_default_device());

//  hipMalloc(&d_x, N*sizeof(double)); 
//  hipMalloc(&d_y, N*sizeof(double));

  status = hipMemGetInfo(&free_bytes, &total_bytes);
  printf("In CUDA C kernel: GPU's memory: %.2f MB used, %.2f MB free.\n", (double)(total_bytes-free_bytes)/1048576.0, (double)free_bytes/1048576.0);

  for (int i = 0; i < N; i++) {
    x[i] = 1.0;
    y[i] = 2.0;
  }

  hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  daxpy_cudac<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(double), hipMemcpyDeviceToHost);

  double maxError = 0.0;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0));
  printf("-- Ran CUDA C kernel.  Max error: %f\n", maxError);

//  hipFree(d_x);
//  hipFree(d_y);

  omp_target_free(d_x, omp_get_default_device());
  omp_target_free(d_y, omp_get_default_device());

  free(x);
  free(y);
}
